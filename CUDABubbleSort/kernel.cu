#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <hip/device_functions.h>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <chrono>
#include <atomic>
#include <thread>
#include <hip/hip_runtime_api.h>

using namespace std;

#define SIZE 1024 // FOR PARALEL GPU  SIZE HAS TO BE 2^n 1024
#define THREADS 4 // FOR PARALEL GPU  THREADS = SIZE / (BLOCKS * 2) 
#define BLOCKS 128//FOR PARALEL GPU  BLOCKS = SIZE / (THREADS * 2) 
string type = "DEVICE";  // USE "HOST" FOR CPU BUBBLE SORT, USE "DEVICE" FOR GPU BUBBLE SORT
int flag = 0;


__host__ void bubbleSortHost(int *array, int index)
{
	int temp;
	do {

		for (int i = 0; i < SIZE - 1 - index * 2 - flag; i++) {
			if (array[index * 2 + i] > array[index * 2 + 1 + i]) {
				temp = array[index * 2 + 1 + i];
				array[index * 2 + 1 + i] = array[index * 2 + i];
				array[index * 2 + i] = temp;
			}
		}

		flag++;

	} while (SIZE - 1 - index * 2 - flag> 0);
}

// NOT USED BUT THERE JUST IN CASE
__global__ void bubbleSortDeviceSerial(int size, int *array)
{
	int i, j, temp;
	for (i = 1; i < size; i++) {
		for (j = 0; j < size - 1; j++) {
			if (array[j] > array[j + 1]) {
				temp = array[j + 1];
				array[j + 1] = array[j];
				array[j] = temp;
			}
		}
	}
}

__global__ void bubbleSortDeviceParallel(int *array, int offSet)
{

	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int indexPerBlock = threadIdx.x;
	int temp;

	if (index  < THREADS* BLOCKS) {

		// FIRST STEP
		if (offSet == 0) {

			// DO THREAD SORTING IN CORRESPONDING BLOCK 
			for (int j = 0; j < THREADS / 2; j++) {

				for (int i = 0; i < THREADS * 2 - 1 - indexPerBlock * 2; i++) {

					if (array[index * 2 + i] > array[index * 2 + 1 + i]) {
						temp = array[index * 2 + 1 + i];
						array[index * 2 + 1 + i] = array[index * 2 + i];
						array[index * 2 + i] = temp;
					}
				}
				__syncthreads();
			}
		}
		// ALL OTHER STEPS, INDEX/THREADS/BLOCKS SHIFTED FOR int offSet
		// LAST BLOCK SKIPPED
		else {
			if (blockIdx.x != BLOCKS - 1) {
				for (int j = 0; j < THREADS / 2; j++) {
					for (int i = offSet; i < THREADS * 2 - 1 + offSet - indexPerBlock * 2; i++) {

						if (array[index * 2 + i] > array[index * 2 + 1 + i]) {
							temp = array[index * 2 + 1 + i];
							array[index * 2 + 1 + i] = array[index * 2 + i];
							array[index * 2 + i] = temp;
						}

					}
					__syncthreads();
				}
			}
		}
	}

}

int main()
{
	srand(time(NULL));
	int h_count = SIZE;
	int counter = BLOCKS;
	int *h_array;
	int *d_array;
	int offSet;

	h_array = new int[h_count];

	// 1. OPTION: TYPE ELEMENTS OF h_array
	/*
	cout << "TYPE ELEMENTS OF ARRAY: " << endl;
	for (int i = 0; i < h_count; i++) {
	cout << i + 1 << ". ELEMENT: ";
	cin >> h_array[i];
	}
	cout << "" << endl;
	*/


	// 2.OPTION: GENERATING RANDOM ELEMENTS FOR h_array
	/*
	for (int i = 0; i < h_count; i++) {
	h_array[i] = rand() % SIZE;
	}
	*/


	// 3. OPTION 999.... 0..1
	for (int i = 0; i < h_count; i++) {
		h_array[i] = SIZE - i;
	}

	// BUBBLE SORT USING CPU
	if (type == "HOST") {

		cout << "ELEMENTS OF ARRAY BEFORE SORT: " << endl;
		for (int i = 0; i < SIZE; i++)
		{
			cout << h_array[i] << " ";
		}
		cout << endl;

		hipEvent_t beginEvent;
		hipEvent_t endEvent;

		hipEventCreate(&beginEvent);
		hipEventCreate(&endEvent);

		hipEventRecord(beginEvent);

		thread bubbleSortCPU[THREADS];
		for (int i = 0; i < THREADS; i++) {
			bubbleSortCPU[i] = thread(bubbleSortHost, h_array, i);
			bubbleSortCPU[i].join();
		}

		hipEventRecord(endEvent);
		hipEventSynchronize(endEvent);

		float timeValue = 0;
		hipEventElapsedTime(&timeValue, beginEvent, endEvent);

		cout << "CPU Time: " << timeValue << endl;
		hipEventDestroy(beginEvent);
		hipEventDestroy(endEvent);

		// ARRAY AFTER BUBBLE SORT
		cout << "BUBBLE SORT RESULTS: " << endl;
		for (int i = 0; i < h_count; i++) {
			cout << h_array[i] << " ";
		}
		cout << endl;
	}

	// BUBBLE SORT USING GPU
	if (type == "DEVICE") {

		cout << "ELEMENTS OF ARRAY BEFORE SORT: " << endl;
		for (int i = 0; i < SIZE; i++)
		{
			cout << h_array[i] << " ";
		}
		cout << endl;

		if (hipMalloc(&d_array, sizeof(int) * h_count) != hipSuccess)
		{
			cout << "D_ARRAY ALLOCATING NOT WORKING!" << endl;
			return 0;
		}

		if (hipMemcpy(d_array, h_array, sizeof(int)* h_count, hipMemcpyHostToDevice) != hipSuccess)
		{
			cout << "hipMemcpyHostToDevice ERROR!" << endl;
			hipFree(d_array);
			return 0;
		}

		hipEvent_t beginEvent;
		hipEvent_t endEvent;

		hipEventCreate(&beginEvent);
		hipEventCreate(&endEvent);

		hipEventRecord(beginEvent);

		do {

			for (int i = 0; i < THREADS * 2; i++) {
				offSet = i;
				// POSSIBLE CHANGE: if offset != 0 USE bubbleSortDeviceParallel << < BLOCKS-1, THREADS >> > (d_array, offSet);
				bubbleSortDeviceParallel << < BLOCKS, THREADS >> > (d_array, offSet);
			}

			counter--;
		} while (counter > 0);

		hipDeviceSynchronize();
		hipEventRecord(endEvent);
		hipEventSynchronize(endEvent);

		float timeValue = 0;
		hipEventElapsedTime(&timeValue, beginEvent, endEvent);

		cout << "GPU Time: " << timeValue << endl;
		hipEventDestroy(beginEvent);
		hipEventDestroy(endEvent);

		if (hipMemcpy(h_array, d_array, sizeof(int)* h_count, hipMemcpyDeviceToHost) != hipSuccess)
		{
			delete[] h_array;
			hipFree(d_array);
			cout << "hipMemcpyDeviceToHost Error" << endl;
			system("pause");
			return 0;
		}
		cout << endl;

		// ARRAY AFTER BUBBLE SORT
		cout << "BUBBLE SORT RESULTS: " << endl;
		for (int i = 0; i < h_count; i++) {
			cout << h_array[i] << " ";
		}
		cout << endl;


	}

	// FREEING MEMORY OF CPU & GPU
	delete[] h_array;
	hipFree(d_array);
	hipDeviceReset();

	system("pause");
	return 0;
}
